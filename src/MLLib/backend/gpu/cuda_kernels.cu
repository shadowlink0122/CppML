/**
 * @file cuda_kernels.cu
 * @brief CUDA kernel implementations for GPU backend
 */

#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <string>

namespace MLLib {
namespace Backend {
namespace cuda {

// Error checking macro
#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      throw std::runtime_error("CUDA error: " +                                \
                               std::string(hipGetErrorString(error)));        \
    }                                                                          \
  } while (0)

#define CUBLAS_CHECK(call)                                                     \
  do {                                                                         \
    hipblasStatus_t status = call;                                              \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
      throw std::runtime_error("cuBLAS error: " + std::to_string(status));     \
    }                                                                          \
  } while (0)

// Thread block size for kernels
const int BLOCK_SIZE = 256;
const int TILE_SIZE = 16;

// CUDA kernel for element-wise addition
__global__ void add_kernel(const double* a, const double* b, double* result,
                           size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] + b[idx];
  }
}

// CUDA kernel for element-wise subtraction
__global__ void subtract_kernel(const double* a, const double* b,
                                double* result, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] - b[idx];
  }
}

// CUDA kernel for element-wise multiplication
__global__ void multiply_kernel(const double* a, const double* b,
                                double* result, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] * b[idx];
  }
}

// CUDA kernel for scalar addition
__global__ void add_scalar_kernel(const double* a, double scalar,
                                  double* result, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] + scalar;
  }
}

// CUDA kernel for scalar multiplication
__global__ void multiply_scalar_kernel(const double* a, double scalar,
                                       double* result, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] * scalar;
  }
}

// CUDA kernel for filling array with value
__global__ void fill_kernel(double* array, double value, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    array[idx] = value;
  }
}

// Optimized matrix multiplication kernel using shared memory
__global__ void matmul_kernel(const double* a, const double* b, double* c,
                              int m, int n, int k) {
  __shared__ double tile_a[TILE_SIZE][TILE_SIZE];
  __shared__ double tile_b[TILE_SIZE][TILE_SIZE];

  int row = blockIdx.y * TILE_SIZE + threadIdx.y;
  int col = blockIdx.x * TILE_SIZE + threadIdx.x;

  double sum = 0.0;

  for (int tile = 0; tile < (k + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
    // Load tiles into shared memory
    if (row < m && tile * TILE_SIZE + threadIdx.x < k) {
      tile_a[threadIdx.y][threadIdx.x] =
          a[row * k + tile * TILE_SIZE + threadIdx.x];
    } else {
      tile_a[threadIdx.y][threadIdx.x] = 0.0;
    }

    if (col < n && tile * TILE_SIZE + threadIdx.y < k) {
      tile_b[threadIdx.y][threadIdx.x] =
          b[(tile * TILE_SIZE + threadIdx.y) * n + col];
    } else {
      tile_b[threadIdx.y][threadIdx.x] = 0.0;
    }

    __syncthreads();

    // Compute partial dot product
    for (int i = 0; i < TILE_SIZE; ++i) {
      sum += tile_a[threadIdx.y][i] * tile_b[i][threadIdx.x];
    }

    __syncthreads();
  }

  if (row < m && col < n) {
    c[row * n + col] = sum;
  }
}

// Global cuBLAS handle
static hipblasHandle_t cublas_handle = nullptr;

// Initialize CUDA context and cuBLAS
void cuda_init() {
  if (cublas_handle == nullptr) {
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));

    // Check if we're in simulation mode
    const char* sim_mode = std::getenv("GPU_SIMULATION_MODE");
    if (sim_mode && std::string(sim_mode) == "1") {
      printf("CUDA simulation mode initialized successfully\n");
    } else {
      printf("CUDA context initialized successfully\n");
    }
  }
}

// Cleanup CUDA context
void cuda_cleanup() {
  if (cublas_handle != nullptr) {
    hipblasDestroy(cublas_handle);
    cublas_handle = nullptr;
  }
}

// Check if CUDA is available
bool cuda_is_available() {
  // Check for GPU simulation mode
  const char* sim_mode = std::getenv("GPU_SIMULATION_MODE");
  if (sim_mode && std::string(sim_mode) == "1") {
    return true;  // Force GPU availability in simulation mode
  }

  int device_count;
  hipError_t error = hipGetDeviceCount(&device_count);
  return (error == hipSuccess && device_count > 0);
}

// Get GPU memory info
void cuda_get_memory_info(size_t* free_bytes, size_t* total_bytes) {
  CUDA_CHECK(hipMemGetInfo(free_bytes, total_bytes));
}

// GPU matrix multiplication using cuBLAS
void cuda_matmul(const double* h_a, const double* h_b, double* h_c, int m,
                 int n, int k) {
  cuda_init();

  size_t size_a = m * k * sizeof(double);
  size_t size_b = k * n * sizeof(double);
  size_t size_c = m * n * sizeof(double);

  double* d_a = nullptr;
  double* d_b = nullptr;
  double* d_c = nullptr;

  try {
    // Allocate GPU memory
    CUDA_CHECK(hipMalloc(&d_a, size_a));
    CUDA_CHECK(hipMalloc(&d_b, size_b));
    CUDA_CHECK(hipMalloc(&d_c, size_c));

    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice));

    // Perform matrix multiplication using cuBLAS
    const double alpha = 1.0, beta = 0.0;
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                             &alpha, d_b, n, d_a, k, &beta, d_c, n));

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost));

  } catch (...) {
    // Cleanup on error
    if (d_a) hipFree(d_a);
    if (d_b) hipFree(d_b);
    if (d_c) hipFree(d_c);
    throw;
  }

  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

// GPU element-wise addition
void cuda_add(const double* h_a, const double* h_b, double* h_result,
              size_t size) {
  double* d_a = nullptr;
  double* d_b = nullptr;
  double* d_result = nullptr;

  size_t byte_size = size * sizeof(double);

  try {
    // Allocate GPU memory
    CUDA_CHECK(hipMalloc(&d_a, byte_size));
    CUDA_CHECK(hipMalloc(&d_b, byte_size));
    CUDA_CHECK(hipMalloc(&d_result, byte_size));

    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));

    // Launch kernel
    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    add_kernel<<<grid_size, BLOCK_SIZE>>>(d_a, d_b, d_result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    CUDA_CHECK(
        hipMemcpy(h_result, d_result, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_a) hipFree(d_a);
    if (d_b) hipFree(d_b);
    if (d_result) hipFree(d_result);
    throw;
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
}

// GPU element-wise subtraction
void cuda_subtract(const double* h_a, const double* h_b, double* h_result,
                   size_t size) {
  double* d_a = nullptr;
  double* d_b = nullptr;
  double* d_result = nullptr;

  size_t byte_size = size * sizeof(double);

  try {
    CUDA_CHECK(hipMalloc(&d_a, byte_size));
    CUDA_CHECK(hipMalloc(&d_b, byte_size));
    CUDA_CHECK(hipMalloc(&d_result, byte_size));

    CUDA_CHECK(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));

    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    subtract_kernel<<<grid_size, BLOCK_SIZE>>>(d_a, d_b, d_result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(
        hipMemcpy(h_result, d_result, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_a) hipFree(d_a);
    if (d_b) hipFree(d_b);
    if (d_result) hipFree(d_result);
    throw;
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
}

// GPU element-wise multiplication
void cuda_multiply(const double* h_a, const double* h_b, double* h_result,
                   size_t size) {
  double* d_a = nullptr;
  double* d_b = nullptr;
  double* d_result = nullptr;

  size_t byte_size = size * sizeof(double);

  try {
    CUDA_CHECK(hipMalloc(&d_a, byte_size));
    CUDA_CHECK(hipMalloc(&d_b, byte_size));
    CUDA_CHECK(hipMalloc(&d_result, byte_size));

    CUDA_CHECK(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice));

    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    multiply_kernel<<<grid_size, BLOCK_SIZE>>>(d_a, d_b, d_result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(
        hipMemcpy(h_result, d_result, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_a) hipFree(d_a);
    if (d_b) hipFree(d_b);
    if (d_result) hipFree(d_result);
    throw;
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
}

// GPU scalar addition
void cuda_add_scalar(const double* h_a, double scalar, double* h_result,
                     size_t size) {
  double* d_a = nullptr;
  double* d_result = nullptr;

  size_t byte_size = size * sizeof(double);

  try {
    CUDA_CHECK(hipMalloc(&d_a, byte_size));
    CUDA_CHECK(hipMalloc(&d_result, byte_size));

    CUDA_CHECK(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));

    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    add_scalar_kernel<<<grid_size, BLOCK_SIZE>>>(d_a, scalar, d_result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(
        hipMemcpy(h_result, d_result, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_a) hipFree(d_a);
    if (d_result) hipFree(d_result);
    throw;
  }

  hipFree(d_a);
  hipFree(d_result);
}

// GPU scalar multiplication
void cuda_multiply_scalar(const double* h_a, double scalar, double* h_result,
                          size_t size) {
  double* d_a = nullptr;
  double* d_result = nullptr;

  size_t byte_size = size * sizeof(double);

  try {
    CUDA_CHECK(hipMalloc(&d_a, byte_size));
    CUDA_CHECK(hipMalloc(&d_result, byte_size));

    CUDA_CHECK(hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice));

    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    multiply_scalar_kernel<<<grid_size, BLOCK_SIZE>>>(d_a, scalar, d_result,
                                                      size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(
        hipMemcpy(h_result, d_result, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_a) hipFree(d_a);
    if (d_result) hipFree(d_result);
    throw;
  }

  hipFree(d_a);
  hipFree(d_result);
}

// GPU fill array
void cuda_fill(double* h_array, double value, size_t size) {
  double* d_array = nullptr;
  size_t byte_size = size * sizeof(double);

  try {
    CUDA_CHECK(hipMalloc(&d_array, byte_size));

    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    fill_kernel<<<grid_size, BLOCK_SIZE>>>(d_array, value, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_array, d_array, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_array) hipFree(d_array);
    throw;
  }

  hipFree(d_array);
}

// GPU copy array
void cuda_copy(const double* h_src, double* h_dst, size_t size) {
  size_t byte_size = size * sizeof(double);

  // For simple copy, we can use cudaMemcpy directly
  // or use GPU kernel for consistency
  double* d_src = nullptr;
  double* d_dst = nullptr;

  try {
    CUDA_CHECK(hipMalloc(&d_src, byte_size));
    CUDA_CHECK(hipMalloc(&d_dst, byte_size));

    CUDA_CHECK(hipMemcpy(d_src, h_src, byte_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dst, d_src, byte_size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(h_dst, d_dst, byte_size, hipMemcpyDeviceToHost));

  } catch (...) {
    if (d_src) hipFree(d_src);
    if (d_dst) hipFree(d_dst);
    throw;
  }

  hipFree(d_src);
  hipFree(d_dst);
}

}  // namespace cuda
}  // namespace Backend
}  // namespace MLLib
